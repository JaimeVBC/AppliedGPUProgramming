#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "macros.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
 */

 /*
  * M = # of rows
  * N = # of columns
  */
int M = 16;
int N = 16;

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, float** outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float* A = (float*)malloc(sizeof(float) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            float* curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0f;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0f)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

void print_partial_matrix(float* M, int nrows, int ncols, int max_row,
    int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}

int main(int argc, char** argv)
{
    float* A, * dA;
    float* B, * dB;
    float* C, * dC;

    int* dANnzPerRow; //Number of elements per row that are not zero
    float* dCsrValA; // Vector with non-zero elements
    int* dCsrRowPtrA; // por d�nde va el puntero del vector
    int* dCsrColIndA; //columna en la que est� cada elemento
    int totalANnz = 0; //valores no cero
    float alpha = 3.0f;
    float beta = 4.0f;
    hipsparseHandle_t handle = NULL;
    hipsparseMatDescr_t Adescr = NULL;
    hipsparseMatDescr_t Bdescr = NULL;
    hipsparseMatDescr_t Cdescr = NULL;

    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(M, N, &A);
    int trueBNnz = generate_random_dense_matrix(N, M, &B);
    C = (float*)malloc(sizeof(float) * M * M);

    printf("A:\n");
    print_partial_matrix(A, M, N, 10, 10);
    printf("B:\n");
    print_partial_matrix(B, N, M, 10, 10);

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate device memory for vectors and the dense form of the matrix A
    CHECK(hipMalloc((void**)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void**)&dB, sizeof(float) * N * M));
    CHECK(hipMalloc((void**)&dC, sizeof(float) * M * M));
    CHECK(hipMalloc((void**)&dANnzPerRow, sizeof(int) * M));

   
    // Construct a descriptor of the matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr( &Adescr));
    CHECK_CUSPARSE(hipsparseSetMatType( Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL)); // redundant because it's the default value
    CHECK_CUSPARSE(hipsparseSetMatIndexBase( Adescr, HIPSPARSE_INDEX_BASE_ZERO)); // redundant because it's the default value

    // Construct a descriptor of the matrix B
    CHECK_CUSPARSE(hipsparseCreateMatDescr( &Bdescr));
    CHECK_CUSPARSE(hipsparseSetMatType( Bdescr, HIPSPARSE_MATRIX_TYPE_GENERAL)); // redundant because it's the default value
    CHECK_CUSPARSE(hipsparseSetMatIndexBase( Bdescr, HIPSPARSE_INDEX_BASE_ZERO)); // redundant because it's the default value

    // Construct a descriptor of the matrix C
    CHECK_CUSPARSE(hipsparseCreateMatDescr( &Cdescr));
    CHECK_CUSPARSE(hipsparseSetMatType( Cdescr, HIPSPARSE_MATRIX_TYPE_GENERAL)); // redundant because it's the default value
    CHECK_CUSPARSE(hipsparseSetMatIndexBase( Cdescr, HIPSPARSE_INDEX_BASE_ZERO)); // redundant because it's the default value

    // Initialize the dense matrix B descriptor
    CHECK_CUSPARSE(hipsparseCreateDnMat((hipsparseDnMatDescr_t*)&Bdescr, M, N, M, B, HIP_R_32F, HIPSPARSE_ORDER_COL));

    // Initialize the dense matrix C descriptor
    CHECK_CUSPARSE(hipsparseCreateDnMat((hipsparseDnMatDescr_t*)&Cdescr, M, N, M, C, HIP_R_32F, HIPSPARSE_ORDER_COL));

    // Transfer the input vectors and dense matrix A to the device
    CHECK(hipMemcpy(dA, A, sizeof(float) * M * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, sizeof(float) * N * M, hipMemcpyHostToDevice));
    CHECK(hipMemset(dC, 0, sizeof(float) * M * M));

    // Compute the number of non-zero elements in A. Total ammount and array with NZ per row
    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, Adescr,
        dA, M, dANnzPerRow, &totalANnz));

    if (totalANnz != trueANnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
            "value: expected %d but got %d\n", trueANnz, totalANnz);
        return 1;
    }

    // Allocate device memory to store the sparse CSR representation of A
    CHECK(hipMalloc((void**)&dCsrValA, sizeof(float) * totalANnz));
    CHECK(hipMalloc((void**)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void**)&dCsrColIndA, sizeof(int) * totalANnz));

    // Convert A from a dense formatting to a CSR formatting, using the GPU
    CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, N, (hipsparseMatDescr_t) Adescr, dA, M, dANnzPerRow,
        dCsrValA, dCsrRowPtrA, dCsrColIndA));
     
    //Create bufferSize
    size_t buffersize;

    //Construct a sparse descriptor of the matrix A
    hipsparseSpMatDescr_t ASpdescr = NULL;
    CHECK_CUSPARSE(hipsparseCreateCsr(&ASpdescr, M, N, totalANnz, dCsrRowPtrA, dCsrColIndA, dCsrValA, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    
    //Get the buffer size for the workspace
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha,  ASpdescr, (hipsparseDnMatDescr_t)Bdescr, &beta, (hipsparseDnMatDescr_t)Cdescr, HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG1, &buffersize));

    //Create buffer
    void* buffer;
    hipMalloc((void**)&buffer, buffersize);
    
    // Perform matrix-matrix multiplication with the CSR-formatted matrix A
    CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, ASpdescr, (hipsparseDnMatDescr_t)Bdescr, &beta, (hipsparseDnMatDescr_t)Cdescr, HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG1, buffer));
    
    // Copy the result vector back to the host
    CHECK(hipMemcpy(C, dC, sizeof(float) * M * M, hipMemcpyDeviceToHost));

    printf("C:\n");
    print_partial_matrix(C, M, M, 10, 10);

    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipFree(dANnzPerRow));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));

    CHECK_CUSPARSE(hipsparseDestroySpMat(ASpdescr));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Bdescr));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Cdescr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
