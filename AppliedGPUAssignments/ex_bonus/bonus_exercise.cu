
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define SEED 921

#define TPB 32

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif


__global__ void kernelMonteCarlo(hiprandState *states, int *d_count, int num_iter_per_thread, int num_threads)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < num_threads)
    {
        double x, y, z;

        int seed = id; // different seed per thread
        hiprand_init(seed, id, 0, &states[id]); // Initialize CURAND
        int cont = 0;
        for (int i = 0; i < num_iter_per_thread; i++)
        {
            x = hiprand_uniform(&states[id]);
            y = hiprand_uniform(&states[id]);
            z = sqrt(x * x + y * y);
            if (z <= 1.0)
            {
                cont++;
            }
        }
        d_count[id] = cont;
    }
}


int main(int argc, char* argv[])
{
    printf("\nLet´s run bonus exercise! \n\n");
    int NUM_ITER = 16384*16;
    float pi = 0.0f;
    int num_iter_per_thread = 32;
    int num_blocks = (NUM_ITER + TPB - 1)/TPB;
    int num_threads = TPB * num_blocks / num_iter_per_thread;
    
    
    printf("\nNUM_ITER:  %d ", NUM_ITER);
    printf("\nNumIterPerThread:  %d ", num_iter_per_thread);
    printf("\nNumThreads:  %d ", num_threads);
    printf("\nThreads per block:  %d ", TPB);
    printf("\nNumber of thread blocks:  %d \n\n", num_blocks);

    srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!

    hiprandState* dev_random;
    hipMalloc((void**)&dev_random, num_threads * sizeof(hiprandState));

    int* d_counts; 
    hipMalloc(&d_counts, num_threads * sizeof(int));

    int* counts;
    counts = (int*) malloc(num_threads * sizeof(int));
    if (counts == NULL) { printf("ERROR! Failure when allocating dynamic memory"); return 1; }

    kernelMonteCarlo KERNEL_ARGS2((num_threads+TPB-1)/TPB, TPB)(dev_random, d_counts, num_iter_per_thread,num_threads);
    hipDeviceSynchronize();
    hipMemcpy(counts, d_counts, num_threads * sizeof(int), hipMemcpyDeviceToHost);


    printf("GPU has finished!\n\n");

    

    int accumulate_count = 0;
    for (int i = 0; i < num_threads; i++)
    {
        /*printf("i vale: %d\n", i);
        printf("counts[i] vale: %d\n", counts[i]);
        printf("accumulate_count vale: %d\n\n", accumulate_count);*/
        accumulate_count += counts[i];
    }
    pi = (float) accumulate_count / NUM_ITER * 4;

    printf("The approximate result of PI is: %lf\n", pi);

    free(counts);
    hipFree(d_counts);
    hipFree(dev_random);

    printf("\nBonus exercise completed! \n\n");

    return 0;
}
